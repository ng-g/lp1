#include <stdio.h>
#include "hip/hip_runtime.h"

#include<bits/stdc++.h>
#include <iostream>
using namespace std;
int n;

__global__ void BSearch(int* da,int num,int n)				// kernel function definition
{

	const int tid = blockIdx.x*blockDim.x + threadIdx.x;

         if(da[tid]==num)
		    da[0]=tid;
}
int main()
{
	int num,nb;
	printf("\n Enter number of elements :");   //you can give any large number
	scanf("%d",&n);
	int a[n+1];
	time_t t;
    srand((unsigned)time(&t));
    a[0]=-1;
    for(unsigned i = 1 ; i <= n ; i++)
    {
    	    	    		a[i]=a[i-1]+rand()%n;  //for generating sorted random sequence
    }

    printf("\n\n Generated array\n");
    for(int i=1; i <=n;i++)
    	printf("%d\t ",a[i]);
    printf("\n Enter number to be searched :");
    scanf("%d",&num);
    //allocating number of blocks

    if(n%1024==0)
    	nb=n/1024;
    else
    	nb=n/1024 +1;

    int* da;//GPU parameter
    //int dpos;

    	hipMalloc(&da, (n+1)*sizeof(int));		//assign memory to parameters on GPU

    	hipMemcpy(da, a, (n+1)*sizeof(int), hipMemcpyHostToDevice);		//copy the array from CPU to GPU

    BSearch<<<nb,1024>>>(da,num,n);
int result;
    hipMemcpy(&result, da, sizeof(int), hipMemcpyDeviceToHost);
 if(result==-1)
        	printf("\nElement not found");
    else
    	printf("\nElement found at %d",result);

return 0;
}







/*OUTPUT:-
 *
 *
 * Enter number of elements :5


 Generated array
2	 4	 7	 9	 11
 Enter number to be searched :7

Element found at 91

*/
 
