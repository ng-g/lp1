#include "hip/hip_runtime.h"



#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#define DIVIDER	10000
__global__ void even(int *darr,int n)
{
   int k=blockIdx.x*512+threadIdx.x;
    int t;
     k=k*2; //for even positions
      if(k< n-1)
       {
          if(darr[k]>darr[k+1])
            {  //swap the numbers
               t=darr[k];
                darr[k]=darr[k+1];
                darr[k+1] =t;
             }
       }
}

__global__ void odd(int *darr,int n)
{
   int k=blockIdx.x*512+threadIdx.x;
    int t;
     k=k*2 +1; //for odd positions
      if(k< n-1)
       {
          if(darr[k]>darr[k+1])
            {  //swap the numbers
               t=darr[k];
                darr[k]=darr[k+1];
                darr[k+1] =t;
             }
       }
}
int main()
{
 int *arr,*darr;
 int n,i;
 time_t t;
     srand((unsigned)time(&t));
  printf("\n Enter how many numbers :");
  scanf("%d",&n);
  arr=(int *)malloc(n*sizeof(int));  //for dynamic inputs

   for(i=0; i<n; i++)
	{
		arr[i] = (rand() % DIVIDER) + 1;
        }
  //  printf("\n UNSORTED ARRAY  \n");
  //   for(i=0; i<n; i++)
    //    printf("\t%d",arr[i]);

  hipMalloc(&darr,n*sizeof(int));  //memory allocation in GPU for darr
  hipMemcpy(darr,arr ,n*sizeof(int) ,hipMemcpyHostToDevice); // data transfer from host to GPU

  for(i=0;i<=n/2;i++)
   {
       even<<<n/1024+1,512>>>(darr,n);
       odd<<<n/1024+1,512>>>(darr,n);
   }
hipMemcpy(arr,darr,n*sizeof(int),hipMemcpyDeviceToHost);

printf("\n SORTED ARRAY  \n");
     for(i=0; i<n; i++)
      printf("\t%d",arr[i]);

}







/*OUPUT:-
 *
 *
 *
 *
 Enter how many numbers :4

 SORTED ARRAY
	1957	6378	7439	8530*/


